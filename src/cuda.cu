// modified from the oxnn package

#include "utils.h"
#include<assert.h>

static int oxnn_cuda_cudaStreamCreate(lua_State *L) {
  hipStream_t *stream = (hipStream_t *)lua_newuserdata(L, sizeof(hipStream_t));
  hipError_t err = hipStreamCreate(stream);

  // check for errors
  if (err != hipSuccess) {
    printf("error in hipStreamCreate: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int oxnn_cuda_cudaStreamDestroy(lua_State *L) {
  hipStream_t *stream = (hipStream_t *)lua_touserdata(L, 1);
  hipError_t err = hipStreamDestroy(*stream);

  // check for errors
  if (err != hipSuccess) {
    printf("error in hipStreamDestroy: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int oxnn_cuda_cudaStreamSynchronize(lua_State *L) {
  hipStream_t *stream = (hipStream_t *)lua_touserdata(L, 1);
  hipError_t err = hipStreamSynchronize(*stream);

  // check for errors
  if (err != hipSuccess) {
    printf("error in hipStreamSynchronize: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int oxnn_cuda_cublasGetStream(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCBlasState *blas_state = state->blasState;
  hipblasHandle_t *handle = blas_state->current_handle;

  hipStream_t *stream = (hipStream_t *)lua_newuserdata(L, sizeof(hipStream_t));
  THCublasCheck(hipblasGetStream(*handle, stream));
  return 1;
}

static int oxnn_cuda_cublasSetStream(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCBlasState *blas_state = state->blasState;
  hipblasHandle_t *handle = blas_state->current_handle;

  hipStream_t *stream = (hipStream_t *)lua_touserdata(L, 1);
  THCublasCheck(hipblasSetStream(*handle, *stream));
  return 1;
}

static const struct luaL_Reg oxnn_cuda__[] = {
    {"hipStreamCreate", oxnn_cuda_cudaStreamCreate},
    {"hipStreamDestroy", oxnn_cuda_cudaStreamDestroy},
    {"hipStreamSynchronize", oxnn_cuda_cudaStreamSynchronize},
    {"hipblasSetStream", oxnn_cuda_cublasSetStream},
    {"hipblasGetStream", oxnn_cuda_cublasGetStream},
    {NULL, NULL}};

void oxnn_cuda_init(lua_State *L) {
  lua_getglobal(L, "oxnn");
  luaL_register(L, NULL, oxnn_cuda__);
  lua_pop(L, 1);
}
